#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "average.h"
#include "../image.h"

#define TILE_SIZE 32

__global__ void CudaWeightedAverage(const float amount, const int width, const int height, CIVL::Pixel* input1, CIVL::Pixel* input2, CIVL::Pixel* output);
__global__ void CudaScreenBlend(const int height, const int width, CIVL::Pixel* input1, CIVL::Pixel* input2, CIVL::Pixel* output);

namespace CIVL{ namespace Average {

    Image WeightedAverage(Image image1, Image image2, float amount){
        // Get the width, height, and pixel data of the image
        int width = image1.width;
        int height = image1.height;

        // Allocate memory on the GPU
        Pixel* image1_data = image1.pixels.data();
        Pixel* image2_data = image2.pixels.data();

        Pixel* d_input1;
        Pixel* d_input2;
        Pixel* d_output;

        hipMalloc(&d_input1, width * height * sizeof(Pixel));
        hipMalloc(&d_input2, width * height * sizeof(Pixel));
        hipMalloc(&d_output, width * height * sizeof(Pixel));

        hipMemcpy(d_input1, image1_data, width * height * sizeof(Pixel), hipMemcpyHostToDevice);
        hipMemcpy(d_input2, image2_data, width * height * sizeof(Pixel), hipMemcpyHostToDevice);

        // Calculate the number of blocks and threads
        dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
        dim3 numBlocks((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE);

        // Call the kernel
        CudaWeightedAverage<<<numBlocks, threadsPerBlock>>>(amount, width, height, d_input1, d_input2, d_output);

        // Copy the image data back to the CPU
        hipMemcpy(image1_data, d_output, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

        // Free the memory on the GPU
        hipFree(d_input1);
        hipFree(d_input2);
        hipFree(d_output);

        return image1;
    }

    Image Average(Image image1, Image image2){
        // Call the weighted average function with an amount of 0.5
        return WeightedAverage(image1, image2, 0.5);
    }

    Image ScreenBlend(Image image1, Image image2){
        // Get the width, height, and pixel data of the image
        int width = image1.width;
        int height = image1.height;

        // Allocate memory on the GPU
        Pixel* image1_data = image1.pixels.data();
        Pixel* image2_data = image2.pixels.data();

        Pixel* d_input1;
        Pixel* d_input2;
        Pixel* d_output;

        hipMalloc(&d_input1, width * height * sizeof(Pixel));
        hipMalloc(&d_input2, width * height * sizeof(Pixel));
        hipMalloc(&d_output, width * height * sizeof(Pixel));

        hipMemcpy(d_input1, image1_data, width * height * sizeof(Pixel), hipMemcpyHostToDevice);
        hipMemcpy(d_input2, image2_data, width * height * sizeof(Pixel), hipMemcpyHostToDevice);

        // Calculate the number of blocks and threads
        dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
        dim3 numBlocks((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE);

        // Call the kernel
        CudaScreenBlend<<<numBlocks, threadsPerBlock>>>(width, height, d_input1, d_input2, d_output);

        // Copy the image data back to the CPU
        hipMemcpy(image1_data, d_output, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

        // Free the memory on the GPU
        hipFree(d_input1);
        hipFree(d_input2);
        hipFree(d_output);

        return image1;
    }
}}

/// \brief Cuda function to perform multiplication on two pixels
/// \param pixel The pixel to multiply
/// \param amount The amount to multiply the pixel by
/// \return The multiplied pixel
__device__ CIVL::Pixel multiplyPixel(const CIVL::Pixel& pixel, float amount) {
    // Multiply the pixel by the amount
    CIVL::Pixel outPixel;
    outPixel.r = pixel.r * amount;
    outPixel.g = pixel.g * amount;
    outPixel.b = pixel.b * amount;
    outPixel.a = 255;

    return outPixel;
}

/// \brief Cuda function to perform addition between two pixels
/// \param pixel1 The first pixel
/// \param pixel2 The second pixel
/// \return The sum of the two pixels
__device__ CIVL::Pixel addPixel(const CIVL::Pixel& pixel1, const CIVL::Pixel& pixel2){
    // Add the two pixels together
    CIVL::Pixel outPixel;
    outPixel.r = pixel1.r + pixel2.r;
    outPixel.g = pixel1.g + pixel2.g;
    outPixel.b = pixel1.b + pixel2.b;
    outPixel.a = 255;

    return outPixel;
}

__global__ void CudaScreenBlend(const int width, const int height, CIVL::Pixel* input1, CIVL::Pixel* input2, CIVL::Pixel* output){
    // Get the x and y coordinates of the pixel
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pixelIndex = y * width + x;

    // Calculate the unit values
    float srcR = input1[pixelIndex].r / 255.0f;
    float srcG = input1[pixelIndex].g / 255.0f;
    float srcB = input1[pixelIndex].b / 255.0f;
    float srcA = input1[pixelIndex].a / 255.0f;

    float dstR = input2[pixelIndex].r / 255.0f;
    float dstG = input2[pixelIndex].g / 255.0f;
    float dstB = input2[pixelIndex].b / 255.0f;
    float dstA = input2[pixelIndex].a / 255.0f;

    // Calculate the output values
    float outR = 1.0f - (1.0f - srcR) * (1.0f - dstR);
    float outG = 1.0f - (1.0f - srcG) * (1.0f - dstG);
    float outB = 1.0f - (1.0f - srcB) * (1.0f - dstB);
    float outA = 1.0f - (1.0f - srcA) * (1.0f - dstA);

    // Recast to pixel values
    output[pixelIndex].r = static_cast<unsigned char>(outR * 255.0f);
    output[pixelIndex].g = static_cast<unsigned char>(outG * 255.0f);
    output[pixelIndex].b = static_cast<unsigned char>(outB * 255.0f);
    output[pixelIndex].a = static_cast<unsigned char>(outA * 255.0f);
}

__global__ void CudaWeightedAverage(const float amount, const int width, const int height, CIVL::Pixel* input1, CIVL::Pixel* input2, CIVL::Pixel* output){
    // Get the x and y coordinates of the pixel
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int pixelIndex = y * width + x;

    // Multiply the pixels by the amount
    CIVL::Pixel pixel1 = multiplyPixel(input1[pixelIndex], (1.0f - amount));
    CIVL::Pixel pixel2 = multiplyPixel(input2[pixelIndex], amount);

    // Add the two pixels together
    output[pixelIndex] = addPixel(pixel1, pixel2);
}

